#include "hip/hip_runtime.h"
/*
Author: Hector Piteau (hector.piteau@gmail.com)
VolumeRendering.cu (c) 2023
Desc: Volume rendering algorithms.
Created:  2023-04-13T12:33:22.433Z
Modified: 2023-05-11T22:28:51.324Z
*/

#include <glm/glm.hpp>
#include <glm/gtc/type_ptr.hpp>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <surface_functions.h>

#include <cuda_gl_interop.h>

#include "../utils/hip/hip_runtime_api.h"
#include <>
#include <cmath>

#include "VolumeRendering.cuh"
#include "../model/RayCaster/Ray.h"
#include "SingleRayCaster.cuh"

#include "Utils.cuh"
#include "Common.cuh"
#include "GPUData.cuh"

using namespace glm;


__device__ float tsdfToAlpha(float tsdf, float previousTsdf, float density) {
    if (previousTsdf > tsdf) {
        return (
                       1.0f + exp(-density * previousTsdf)) /
               (1.0f + exp(-density * tsdf));
    } else {
        return 1.0f;
    }
}


__device__ bool IsPointInBBox(const vec3 &point, VolumeDescriptor *volume) {
    if (all(lessThan(point, volume->bboxMax)) && all(greaterThan(point, volume->bboxMin)))
        return true;
    else
        return false;
}

__device__ short IsPointInVolume(const vec3 &point) {
    if (any(lessThan(point, vec3(-0.5, -0.5, -0.5))) || any(greaterThan(point, vec3(0.5, 0.5, 0.5))))
        return 0;
    return 1;
}

__device__ vec4 forward(Ray &ray, VolumeDescriptor *volume) //, float4* volume, const ivec3& resolution)
{
    /** Partial transmittance. */
    float Tpartial = 1.0f;
    /** Partial color. */
    vec3 Cpartial = vec3(0.0f, 0.0f, 0.0f);

    float step = 0.01f;

    /** The ray's min must be strictly smaller than max. */
    if (ray.tmin < ray.tmax) {

        /** Travel through the ray from it's min to max. */
        for (float t = ray.tmin; t < ray.tmax; t += step) {
            vec3 pos = ray.origin + t * ray.dir;

            if (IsPointInBBox(pos, volume)) {
                vec4 data = ReadVolume(pos, volume);
                vec3 color = vec3(data.r, data.g, data.b);
                float alpha = data.a;

                Cpartial += Tpartial * (1 - exp(-alpha)) * color;
//                Cpartial += Tpartial * alpha * color;
//                Tpartial *= (1.0f - alpha);
                Tpartial *= (1.0f / exp(alpha));

                if (Tpartial < 0.001f) {
                    Tpartial = 0.0f;
                    break;
                }
            }
        }
    }
    return {Cpartial, Tpartial};
}


__global__ void volumeRenderingUI8(RayCasterDescriptor *raycaster, CameraDescriptor *camera, VolumeDescriptor *volume,
                                   hipSurfaceObject_t surface) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= camera->width || y >= camera->height) return;

    if (!raycaster->renderAllPixels) {
        uint minpx = camera->width - raycaster->minPixelX;
        uint minpy = camera->height - raycaster->minPixelY;

        uint maxpx = camera->width - raycaster->maxPixelX;
        uint maxpy = camera->height - raycaster->maxPixelY;

        uint4 a = make_uint4(maxpx, maxpy, minpx, minpy);
        minpx = a.x;
        minpy = a.y;
        maxpx = a.z;
        maxpy = a.w;

//        if (x > minpx - 5 && x < minpx + 5 && y > minpy - 5 && y < minpy + 5) {
//            surf2Dwrite<uchar4>(make_uchar4(255, 255, 0, 255), surface, x * sizeof(uchar4), y);
//            return;
//        }
//
//        if (x > maxpx - 5 && x < maxpx + 5 && y > maxpy - 5 && y < maxpy + 5) {
//            surf2Dwrite<uchar4>(make_uchar4(0, 255, 255, 255), surface, x * sizeof(uchar4), y);
//            return;
//        }

        if (x >= minpx && x <= maxpx && y >= minpy && y <= maxpy) {
            Ray ray = SingleRayCaster::GetRay(vec2(camera->width - x, camera->height - y), camera);
            bool res = BBoxTminTmax(ray.origin, ray.dir, volume->bboxMin, volume->bboxMax, &ray.tmin, &ray.tmax);
            if (!res) {
                uchar4 element = make_uchar4(0, 0, 0, 0);
                surf2Dwrite<uchar4>(element, surface, x * sizeof(uchar4), y);
                return;
            }

            /** Call forward. */
            vec4 result = forward(ray, volume) * 255.0f;
            uchar4 element = make_uchar4(result.x, result.y, result.z, 255.0f);
            surf2Dwrite<uchar4>(element, surface, (x) * sizeof(uchar4), y);
        } else {
            uchar4 element = make_uchar4(0, 0, 0, 0);
            surf2Dwrite<uchar4>(element, surface, x * sizeof(uchar4), y);
        }

    } else {
        Ray ray = SingleRayCaster::GetRay(vec2(camera->width - x, camera->height - y), camera);
        /** Call forward. */
        vec4 result = forward(ray, volume) * 255.0f;
        uchar4 element = make_uchar4(result.x, result.y, result.z, result.w);
        surf2Dwrite<uchar4>(element, surface, (x) * sizeof(uchar4), y);
    }
}

extern "C" void volume_rendering_wrapper(GPUData<RayCasterDescriptor> &raycaster, GPUData<CameraDescriptor> &camera,
                                         GPUData<VolumeDescriptor> &volume, hipSurfaceObject_t surface) {
    /** Max 1024 per block. As each pixel is independent, may be useful to search for optimal size. */
    dim3 threadsPerBlock(16, 16);
    /** This create enough blocks to cover the whole texture, may contain threads that does not have pixel's assigned. */
    dim3 numBlocks(
            (camera.Host()->width + threadsPerBlock.x - 1) / threadsPerBlock.x,
            (camera.Host()->height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    /** Call the main volumeRendering kernel. **/
    volumeRenderingUI8<<<numBlocks, threadsPerBlock>>>(raycaster.Device(), camera.Device(), volume.Device(), surface);

    /** Get last error after rendering. */
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "ERROR: " << hipGetErrorString(err) << std::endl;
    }

    hipDeviceSynchronize();
}


__global__ void batched_forward(VolumeDescriptor *volume, BatchItemDescriptor *item) {
    /** Pixel coords. */
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= item->cam->width || y >= item->cam->height) return;

    uchar4 ground_truth = make_uchar4(
            item->img->data[STBI_IMG_INDEX(x, y, item->img->res.x, item->img->res.y)],
            item->img->data[STBI_IMG_INDEX(x, y, item->img->res.x, item->img->res.y) + 1],
            item->img->data[STBI_IMG_INDEX(x, y, item->img->res.x, item->img->res.y) + 2],
            item->img->data[STBI_IMG_INDEX(x, y, item->img->res.x, item->img->res.y) + 3]
    );
    vec3 gt_color = UCHAR4_TO_VEC3(ground_truth);

    Ray ray = SingleRayCaster::GetRay(ivec2(x, y), item->cam);
    ray.tmin = item->range->data[LINEAR_IMG_INDEX(x, y, item->range->dim.y)].x;
    ray.tmax = item->range->data[LINEAR_IMG_INDEX(x, y, item->range->dim.y)].y;

    /** Run forward function. */
    vec4 res = forward(ray, volume);
    item->loss[LINEAR_IMG_INDEX(x, y, item->res.y)] = res;

    /** Store loss. */
    float epsilon = 0.001f;
    vec3 pred_color = vec3(res);
    vec3 loss = (gt_color - pred_color) / (pred_color + epsilon);
    item->loss[LINEAR_IMG_INDEX(x, y, item->res.y)] = loss;


    if (item->debugRender) {
//        uchar4 element = ground_truth;
//        uchar4 element = FLOAT4_NORM_TO_UCHAR4(res);
        uchar4 element = VEC3_255_TO_UCHAR4(loss);
        surf2Dwrite<uchar4>(element, item->debugSurface, (x) * sizeof(uchar4), y);
    }

}

__global__ void batched_backward(VolumeDescriptor *volume, BatchItemDescriptor *item, AdamOptimizerDescriptor* adam) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= item->cam->width || y >= item->cam->height) return;

    Ray ray = SingleRayCaster::GetRay(ivec2(x, y), item->cam);
    ray.tmin = item->range->data[LINEAR_IMG_INDEX(x, y, item->range->dim.y)].x;
    ray.tmax = item->range->data[LINEAR_IMG_INDEX(x, y, item->range->dim.y)].y;

    uchar4 ground_truth = make_uchar4(
            item->img->data[STBI_IMG_INDEX(x, y, item->img->res.x, item->img->res.y)],
            item->img->data[STBI_IMG_INDEX(x, y, item->img->res.x, item->img->res.y) + 1],
            item->img->data[STBI_IMG_INDEX(x, y, item->img->res.x, item->img->res.y) + 2],
            item->img->data[STBI_IMG_INDEX(x, y, item->img->res.x, item->img->res.y) + 3]
    );
    vec3 cgt = UCHAR4_TO_VEC3(ground_truth);

    float epsilon = 0.001f;
    float zeroCross = INFINITY; //0x7f800000; //std::numeric_limits<float>().infinity();
    bool gradWritten = false;

    auto loss = item->loss[LINEAR_IMG_INDEX(x, y, item->res.y)];
    auto cpred = item->cpred[LINEAR_IMG_INDEX(x, y, item->res.y)];

    auto dLdC = (2.0f * (cpred - cgt)) / (cpred + vec3(epsilon));
    dLdC = clamp(dLdC, -10.0f, 10.0f);

    /** Partial transmittance. */
    float Tpartial = 1.0f;
    /** Partial color. */
    vec3 Cpartial = vec3(0.0f, 0.0f, 0.0f);

    float step = 0.01f;

    /** The ray's min must be strictly smaller than max. */
    if (ray.tmin < ray.tmax) {

        /** Travel through the ray from it's min to max. */
        for (float t = ray.tmin; t < ray.tmax; t += step) {
            vec3 pos = ray.origin + t * ray.dir;

            if (IsPointInBBox(pos, volume)) {
                vec4 data = ReadVolume(pos, volume);
                vec3 color = vec3(data.r, data.g, data.b);
                float alpha = data.a;

//                Cpartial += Tpartial * alpha * color;
                Cpartial += Tpartial * (1 - exp(-alpha)) * color;

                /** Compute full loss */
                auto dLo_dCi = Tpartial * ( 1 - exp(-alpha));
                auto color_grad = dLdC * dLo_dCi;

                WriteVolumeTRI(adam->);

//                Tpartial *= (1.0f - alpha);
                Tpartial *= (1.0f / exp(alpha));

                if (Tpartial < 0.001f) {
                    Tpartial = 0.0f;
                    break;
                }
            }
        }
    }


}


extern "C" void batched_backward_wrapper(GPUData<BatchItemDescriptor> &item, GPUData<VolumeDescriptor> &volume) {
    dim3 threads(16, 16);
    /** This create enough blocks to cover the whole texture, may contain threads that does not have pixel's assigned. */
    dim3 blocks((item.Host()->res.x + threads.x - 1) / threads.x,
                (item.Host()->res.y + threads.y - 1) / threads.y);

    batched_backward<<<blocks, threads>>>(volume.Device(), item.Device());
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "(batched_forward_wrapper) ERROR: " << hipGetErrorString(err) << std::endl;
    }
}

extern "C" void batched_forward_wrapper(GPUData<BatchItemDescriptor> &item, GPUData<VolumeDescriptor> &volume) {
    dim3 threads(16, 16);
    /** This create enough blocks to cover the whole texture, may contain threads that does not have pixel's assigned. */
    dim3 blocks((item.Host()->res.x + threads.x - 1) / threads.x,
                (item.Host()->res.y + threads.y - 1) / threads.y);

    batched_forward<<<blocks, threads>>>(volume.Device(), item.Device());
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "(batched_forward_wrapper) ERROR: " << hipGetErrorString(err) << std::endl;
    }
}