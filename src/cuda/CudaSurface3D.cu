#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <surface_functions.h>
#include <iostream>

#include "CudaSurface3D.cuh"
#include "../utils/hip/hip_runtime_api.h"

#define SURFACES_AMOUNT 5

// surface<void, cudaSurfaceType3D> surfaces3d[SURFACES_AMOUNT];

extern "C" void cuda_surface_wrapper()
{
    
}