#include "hip/hip_runtime.h"
/*
Author: Hector Piteau (hector.piteau@gmail.com)
Volume.cu (c) 2023
Desc: Volume algorithms.
Created:  2023-04-13T12:33:22.433Z
Modified: 2023-05-11T22:28:51.324Z
*/

#include <glm/glm.hpp>
#include <glm/gtc/type_ptr.hpp>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <surface_functions.h>

#include <cuda_gl_interop.h>

#include "../utils/hip/hip_runtime_api.h"
#include <>
#include <cmath>

#include "Volume.cuh"
#include "../model/RayCaster/Ray.h"
#include "SingleRayCaster.cuh"

#include "Utils.cuh"
#include "Common.cuh"
#include "GPUData.cuh"

using namespace glm;

//__global__ void volume_resize_double(cell* source_volume, cell* target_volume, const ivec3& source_res, const ivec3& target_res){
//__global__ void volume_resize_double(cell* source_volume, const ivec3& source_res){
//    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
//    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
//    unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;
//
//    if(x > source_res.x || y > source_res.y || z > source_res.z) return;
////    if(x >= target_res.x || y >= target_res.y || z >= target_res.z) return;
//
////    target_volume[VOLUME_INDEX(x, y, z, target_res)].data.x = 1.0f;
////    target_volume[VOLUME_INDEX(x, y, z, target_res)].data.y = 0.0f;
////    target_volume[VOLUME_INDEX(x, y, z, target_res)].data.z = 0.0f;
////    target_volume[VOLUME_INDEX(x, y, z, target_res)].data.w = 1.0f;
//
//    /** For the thread in source, write its value in the target volume. */
////    auto source_cell = source_volume[VOLUME_INDEX(x,y,z, source_res)];
//    auto index = VOLUME_INDEX(x,y,z, source_res);
//    source_volume[index].data = make_float4(1.0, 0.0, 0.0, 1.0);
//
////    ivec3 target_coords = ivec3(x,y,z);
////    if(target_coords.x >= target_res.x || target_coords.y >= target_res.y || target_coords.z >= target_res.z) return;
//
//    /** same coord */
////    int index = VOLUME_INDEX(target_coords.x,target_coords.y,target_coords.z, target_res);
////    target_volume[index].data = make_float4(source_cell.data.x, source_cell.data.y, source_cell.data.z, source_cell.data.w);
////if(x == 2 && y == 2 && z == 2){
////    debug->i = index;
////    debug->x = x;
////    debug->y = y;
////    debug->z = z;
////    debug->iv3 = target_coords;
////
////}
//
////    target_coords = ivec3(x + 1,y,z) * 2;
////    if(target_coords.x > target_res.x || target_coords.y > target_res.y || target_coords.z > target_res.z){
////        /** x+1 */
////        target_volume[VOLUME_INDEX(target_coords.x,target_coords.y,target_coords.z, target_res)] = source_cell;
////    }
////    target_coords = ivec3(x,y,z + 1) * 2;
////    if(target_coords.x > target_res.x || target_coords.y > target_res.y || target_coords.z > target_res.z){
////        /** z+1 */
////        target_volume[VOLUME_INDEX(target_coords.x,target_coords.y,target_coords.z, target_res)] = source_cell;
////    }
////
////    target_coords = ivec3(x + 1,y,z + 1) * 2;
////    if(target_coords.x > target_res.x || target_coords.y > target_res.y || target_coords.z > target_res.z){
////        /** x+1, z+1 */
////        target_volume[VOLUME_INDEX(target_coords.x,target_coords.y,target_coords.z, target_res)] = source_cell;
////    }
////
////    target_coords = ivec3(x + 1, y + 1, z) * 2;
////    if(target_coords.x > target_res.x || target_coords.y > target_res.y || target_coords.z > target_res.z){
////        /** y+1, x+1 */
////        target_volume[VOLUME_INDEX(target_coords.x,target_coords.y,target_coords.z, target_res)] = source_cell;
////    }
////
////    target_coords = ivec3(x, y + 1, z + 1) * 2;
////    if(target_coords.x > target_res.x || target_coords.y > target_res.y || target_coords.z > target_res.z){
////        /** y+1, z+1 */
////        target_volume[VOLUME_INDEX(target_coords.x,target_coords.y,target_coords.z, target_res)] = source_cell;
////    }
////
////    target_coords = ivec3(x + 1, y + 1, z + 1) * 2;
////    if(target_coords.x > target_res.x || target_coords.y > target_res.y || target_coords.z > target_res.z){
////        /** y+1, x+1, z+1 */
////        target_volume[VOLUME_INDEX(target_coords.x,target_coords.y,target_coords.z, target_res)] = source_cell;
////    }
//}


__global__ void volume_resize_double(VolumeDescriptor *source,VolumeDescriptor *target){
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

    if(x > source->res.x || y > source->res.y || z > source->res.z) return;

    /** For the thread in source, write its value in the target volume. */
    auto source_index = VOLUME_INDEX(x,y,z, source->res);
    auto source_cell = source->data[source_index];

    ivec3 target_coords = ivec3(x,y,z) * 2;
    if(target_coords.x >= target->res.x || target_coords.y >= target->res.y || target_coords.z >= target->res.z) return;

    /** same coord */
    int index = VOLUME_INDEX(target_coords.x,target_coords.y,target_coords.z, target->res);
    target->data[index].data = make_float4(source_cell.data.x, source_cell.data.y, source_cell.data.z, source_cell.data.w);

    target_coords = ivec3(x + 1,y,z) * 2;
    if(target_coords.x > target->res.x || target_coords.y > target->res.y || target_coords.z > target->res.z){
        /** x+1 */
        target->data[VOLUME_INDEX(target_coords.x,target_coords.y,target_coords.z, target->res)] = source_cell;
    }
    target_coords = ivec3(x,y,z + 1) * 2;
    if(target_coords.x > target->res.x || target_coords.y > target->res.y || target_coords.z > target->res.z){
        /** z+1 */
        target->data[VOLUME_INDEX(target_coords.x,target_coords.y,target_coords.z, target->res)] = source_cell;
    }

    target_coords = ivec3(x + 1,y,z + 1) * 2;
    if(target_coords.x > target->res.x || target_coords.y > target->res.y || target_coords.z > target->res.z){
        /** x+1, z+1 */
        target->data[VOLUME_INDEX(target_coords.x,target_coords.y,target_coords.z, target->res)] = source_cell;
    }

    target_coords = ivec3(x + 1, y + 1, z) * 2;
    if(target_coords.x > target->res.x || target_coords.y > target->res.y || target_coords.z > target->res.z){
        /** y+1, x+1 */
        target->data[VOLUME_INDEX(target_coords.x,target_coords.y,target_coords.z, target->res)] = source_cell;
    }

    target_coords = ivec3(x, y + 1, z + 1) * 2;
    if(target_coords.x > target->res.x || target_coords.y > target->res.y || target_coords.z > target->res.z){
        /** y+1, z+1 */
        target->data[VOLUME_INDEX(target_coords.x,target_coords.y,target_coords.z, target->res)] = source_cell;
    }

    target_coords = ivec3(x + 1, y + 1, z + 1) * 2;
    if(target_coords.x > target->res.x || target_coords.y > target->res.y || target_coords.z > target->res.z){
        /** y+1, x+1, z+1 */
        target->data[VOLUME_INDEX(target_coords.x,target_coords.y,target_coords.z, target->res)] = source_cell;
    }

}

extern "C" void volume_resize_double_wrapper(GPUData<VolumeDescriptor>& source, GPUData<VolumeDescriptor>& target){
    dim3 threads(8,8,8);
    /** This create enough blocks to cover the whole volume,
     * may contain threads that does not have pixel's assigned. */
    dim3 blocks((source.Host()->res.x + threads.x - 1) / threads.x,
                (source.Host()->res.y + threads.y - 1) / threads.y,
                (source.Host()->res.z + threads.z - 1) / threads.z);

    volume_resize_double<<<blocks, threads>>>(source.Device(), target.Device());
    hipDeviceSynchronize();

    std::cout << "Resize volume done. " << std::endl;

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "(volume_resize_double) ERROR: " << hipGetErrorString(err) << std::endl;
    }
}

